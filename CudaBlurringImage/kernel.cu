#include "hip/hip_runtime.h"
﻿#include <raylib.h>
#include "hip/hip_runtime.h"
#include <iostream>

const int BLUR_SIZE = 20;
const int CHANNELS = 4; // RGBA

__global__ void BlurImageKernel(unsigned char* in, unsigned char* out, int w, int h) {
    int row = threadIdx.y + blockDim.y * blockIdx.y;
    int col = threadIdx.x + blockDim.x * blockIdx.x;

    if (col < w && row < h) {
        for (int c = 0; c < CHANNELS; ++c) {
            int pixVal = 0;
            int pixels = 0;
            for (int blurRow = -BLUR_SIZE; blurRow <= BLUR_SIZE; ++blurRow) {
                for (int blurCol = -BLUR_SIZE; blurCol <= BLUR_SIZE; ++blurCol) {
                    int curRow = row + blurRow;
                    int curCol = col + blurCol;

                    if (curRow >= 0 && curRow < h && curCol >= 0 && curCol < w) {
                        pixVal += in[(curRow * w + curCol) * CHANNELS + c];
                        pixels++;
                    }
                }
            }
            out[(row * w + col) * CHANNELS + c] = pixVal / pixels;
        }
    }
}

int main() {
    // Load the image
    Image goku = LoadImage("C:/Users/ochoa/Desktop/Programing/Cuda/Learning/CudaColorToGrayScale/goku.png");

    if (goku.data == NULL) {
        std::cerr << "Failed to load image" << std::endl;
        return -1;
    }

    // Ensure the image is in the correct format
    ImageFormat(&goku, PIXELFORMAT_UNCOMPRESSED_R8G8B8A8);

    // Get image data
    unsigned char* imgData = (unsigned char*)goku.data;
    int imgWidth = goku.width;
    int imgHeight = goku.height;
    int imgSize = imgWidth * imgHeight * CHANNELS * sizeof(unsigned char);

    // Initialize the window to match the image size
    InitWindow(imgWidth, imgHeight, "Blurring Image");

    // Allocate memory on the device
    unsigned char* d_in, * d_out;
    hipMalloc((void**)&d_in, imgSize);
    hipMalloc((void**)&d_out, imgSize);

    // Copy image data to the device
    hipMemcpy(d_in, imgData, imgSize, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(16, 16, 1);
    dim3 gridSize((imgWidth + blockSize.x - 1) / blockSize.x, (imgHeight + blockSize.y - 1) / blockSize.y, 1);

    // Launch the kernel
    BlurImageKernel << <gridSize, blockSize >> > (d_in, d_out, imgWidth, imgHeight);

    // Allocate memory for the result on the host
    unsigned char* blurredImgData = new unsigned char[imgWidth * imgHeight * CHANNELS];

    // Copy the result back to the host
    hipMemcpy(blurredImgData, d_out, imgSize, hipMemcpyDeviceToHost);

    // Create a new image for the blurred result
    Image blurredImg = {
        blurredImgData,
        imgWidth,
        imgHeight,
        1,
        PIXELFORMAT_UNCOMPRESSED_R8G8B8A8
    };

    // Convert the image to a texture
    Texture2D texture = LoadTextureFromImage(blurredImg);

    // Unload the original image and device memory
    UnloadImage(goku);
    hipFree(d_in);
    hipFree(d_out);

    // Main game loop
    while (!WindowShouldClose()) {
        BeginDrawing();
        ClearBackground(WHITE);

        // Draw the texture
        DrawTexture(texture, 0, 0, WHITE);

        EndDrawing();
    }

    // Unload the texture
    UnloadTexture(texture);

    // Clean up the blurred image data
    delete[] blurredImgData;

    // Close the window and terminate Raylib
    CloseWindow();

    return 0;
}
